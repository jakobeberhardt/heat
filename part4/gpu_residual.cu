
#include <hip/hip_runtime.h>

__global__ void gpu_residual(float *u, float *utmp, float *residual,int N) {
    // Calculate the thread's unique index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = i * N + j;

    // Shared memory for in-block reduction
    extern __shared__ float sdata[];

    // Each thread computes one element (if within the domain boundaries)
    float diff = 0.0;
    if (i > 0 && i < N - 1 && j > 0 && j < N - 1) {
        diff = utmp[index] - u[index];
    }
    sdata[threadIdx.y * blockDim.x + threadIdx.x] = diff * diff;

    __syncthreads(); // Wait for all threads in the block to finish updating shared memory

    // Perform in-block reduction
    // For simplicity, we assume blockDim.x * blockDim.y is a power of 2
    int blockSize = blockDim.x * blockDim.y;
    if (blockSize >= 512 && threadIdx.x < 256) { sdata[threadIdx.y * blockDim.x + threadIdx.x] += sdata[threadIdx.y * blockDim.x + threadIdx.x + 256]; } __syncthreads();
    if (blockSize >= 256 && threadIdx.x < 128) { sdata[threadIdx.y * blockDim.x + threadIdx.x] += sdata[threadIdx.y * blockDim.x + threadIdx.x + 128]; } __syncthreads();
    if (blockSize >= 128 && threadIdx.x <  64) { sdata[threadIdx.y * blockDim.x + threadIdx.x] += sdata[threadIdx.y * blockDim.x + threadIdx.x +  64]; } __syncthreads();

    // Now that we are using 64 threads or less, we can assume that we are within a warp and no longer need to synchronize
    if (threadIdx.x < 32) {
        volatile float* smem = sdata;
        if (blockSize >=  64) { smem[threadIdx.x] += smem[threadIdx.x + 32]; }
        if (blockSize >=  32) { smem[threadIdx.x] += smem[threadIdx.x + 16]; }
        if (blockSize >=  16) { smem[threadIdx.x] += smem[threadIdx.x +  8]; }
        if (blockSize >=   8) { smem[threadIdx.x] += smem[threadIdx.x +  4]; }
        if (blockSize >=   4) { smem[threadIdx.x] += smem[threadIdx.x +  2]; }
        if (blockSize >=   2) { smem[threadIdx.x] += smem[threadIdx.x +  1]; }
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        residual[blockIdx.x + gridDim.x * blockIdx.y] = sdata[0];
    }
}
