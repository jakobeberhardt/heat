#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat(double *h, double *g, int N) {
	int i = blockIdx.y* blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >0 && i < N -1 && j> 0 && j< N-1){
		h[i*N+j]= 0.25 * (g[i * N + (j-1) ]+  // left
					        g[ i * N + (j+1) ]+  // right
				          g[ (i-1) * N + j ]+  // top
				          g[ (i+1) * N + j ]); // bottom
	}
}


__global__ void gpu_Residual(double *u, double *utmp,double *dev_diff, double *residuals, int N){
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int index = i * N + j;
  unsigned int diff_index = (i-1)*(N-2)+j-1;
    if (i > 0 && i < N - 1 && j > 0 && j < N - 1) {
        dev_diff[diff_index] = utmp[index] - u[index];
        residuals[diff_index]=dev_diff[diff_index]*dev_diff[diff_index];
}
}

__global__ void Kernel07(double *g_idata, double *g_odata, int N) {
  __shared__ double sdata[1024];
  unsigned int s;

  // Cada thread realiza la suma parcial de los datos que le
  // corresponden y la deja en la memoria compartida
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int gridSize = blockDim.x*2*gridDim.x;
  sdata[tid] = 0;
  while (i < N) {
    sdata[tid] += g_idata[i] + g_idata[i+blockDim.x];
    i += gridSize;
  }
  __syncthreads();

  // Hacemos la reduccion en la memoria compartida
  for (s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  // desenrrollamos el ultimo warp activo
  if (tid < 32) {
    volatile double *smem = sdata;

    smem[tid] += smem[tid + 32];
    smem[tid] += smem[tid + 16];
    smem[tid] += smem[tid + 8];
    smem[tid] += smem[tid + 4];
    smem[tid] += smem[tid + 2];
    smem[tid] += smem[tid + 1];
  }


  // El thread 0 escribe el resultado de este bloque en la memoria global
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

__global__ void finalReduceKernel(double *g_idata, double *g_odata, int N) {
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;

    // Load block sums from global memory to shared memory
    sdata[tid] = (tid < N) ? g_idata[tid] : 0;
    __syncthreads();

    // Perform final reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the final result to global memory
    if (tid == 0) g_odata[0] = sdata[0];
}