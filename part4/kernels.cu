#include <math.h>
#include <float.h>
#include <hip/hip_runtime.h>

__global__ void gpu_Heat (double *h, double *g, int N) {
	int i = blockIdx.y* blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >0 && i < N -1 && j> 0 && j< N-1){
		h[i*N+j]= 0.25 * (g[i * N + (j-1) ]+  // left
					      g[ i * N + (j+1) ]+  // right
				          g[ (i-1) * N + j ]+  // top
				          g[ (i+1) * N + j ]); // bottom
	}
}

__global__ void gpu_residual(double *u, double *utmp, double *residual, int N) {
  // Calculate the thread's unique index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = i * N + j;

    // Shared memory for in-block reduction
    extern __shared__ double sdata[];

    // Each thread computes one element (if within the domain boundaries)
    double diff = 0.0;
    if (i > 0 && i < N - 1 && j > 0 && j < N - 1) {
        diff = utmp[index] - u[index];
    }
    sdata[threadIdx.y * blockDim.x + threadIdx.x] = diff * diff;

    __syncthreads(); // Wait for all threads in the block to finish updating shared memory

    // Perform in-block reduction
    // For simplicity, we assume blockDim.x * blockDim.y is a power of 2
    int blockSize = blockDim.x * blockDim.y;
    if (blockSize >= 512 && threadIdx.x < 256) { sdata[threadIdx.y * blockDim.x + threadIdx.x] += sdata[threadIdx.y * blockDim.x + threadIdx.x + 256]; } __syncthreads();
    if (blockSize >= 256 && threadIdx.x < 128) { sdata[threadIdx.y * blockDim.x + threadIdx.x] += sdata[threadIdx.y * blockDim.x + threadIdx.x + 128]; } __syncthreads();
    if (blockSize >= 128 && threadIdx.x <  64) { sdata[threadIdx.y * blockDim.x + threadIdx.x] += sdata[threadIdx.y * blockDim.x + threadIdx.x +  64]; } __syncthreads();

    // Now that we are using 64 threads or less, we can assume that we are within a warp and no longer need to synchronize
    if (threadIdx.x < 32) {
        volatile double* smem = sdata;
        if (blockSize >=  64) { smem[threadIdx.x] += smem[threadIdx.x + 32]; }
        if (blockSize >=  32) { smem[threadIdx.x] += smem[threadIdx.x + 16]; }
        if (blockSize >=  16) { smem[threadIdx.x] += smem[threadIdx.x +  8]; }
        if (blockSize >=   8) { smem[threadIdx.x] += smem[threadIdx.x +  4]; }
        if (blockSize >=   4) { smem[threadIdx.x] += smem[threadIdx.x +  2]; }
        if (blockSize >=   2) { smem[threadIdx.x] += smem[threadIdx.x +  1]; }
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        residual[blockIdx.x + gridDim.x * blockIdx.y] = sdata[0];
    }
}

__global__ void gpu_Residual(double *u, double *utmp,double *dev_diff, double *residuals, int N){
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int index = i * N + j;
  unsigned int diff_index = (i-1)*(N-2)+j-1;
    if (i > 0 && i < N - 1 && j > 0 && j < N - 1) {
        dev_diff[diff_index] = utmp[index] - u[index];
        residuals[diff_index]=dev_diff[diff_index]*dev_diff[diff_index];
}
}

__global__ void Kernel07(double *g_idata, double *g_odata, int N) {
  __shared__ double sdata[1024];
  unsigned int s;

  // Cada thread realiza la suma parcial de los datos que le
  // corresponden y la deja en la memoria compartida
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  unsigned int gridSize = blockDim.x*2*gridDim.x;
  sdata[tid] = 0;
  while (i < N) {
    sdata[tid] += g_idata[i] + g_idata[i+blockDim.x];
    i += gridSize;
  }
  __syncthreads();

  // Hacemos la reduccion en la memoria compartida
  for (s=blockDim.x/2; s>32; s>>=1) {
    if (tid < s)
      sdata[tid] += sdata[tid + s];
    __syncthreads();
  }
  // desenrrollamos el ultimo warp activo
  if (tid < 32) {
    volatile double *smem = sdata;

    smem[tid] += smem[tid + 32];
    smem[tid] += smem[tid + 16];
    smem[tid] += smem[tid + 8];
    smem[tid] += smem[tid + 4];
    smem[tid] += smem[tid + 2];
    smem[tid] += smem[tid + 1];
  }


  // El thread 0 escribe el resultado de este bloque en la memoria global
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];

}

__global__ void finalReduceKernel(double *g_idata, double *g_odata, int N) {
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;

    // Load block sums from global memory to shared memory
    sdata[tid] = (tid < N) ? g_idata[tid] : 0;
    __syncthreads();

    // Perform final reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the final result to global memory
    if (tid == 0) g_odata[0] = sdata[0];
}